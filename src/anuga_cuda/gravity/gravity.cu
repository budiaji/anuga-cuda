
#include <hip/hip_runtime.h>
//#define REARRANGED_DOMAIN 

#ifdef USING_SHARED_MEMORY
#define BLOCK_SIZE 960
#endif

__global__ void gravity_wb(
        int N,
        double g,
        double * stage_vertex_values, 
        double * stage_edge_values, 
        double * stage_centroid_values, 
        double * bed_edge_values, 
        double * bed_centroid_values, 
        double * vertex_coordinates, 
        double * xmom_explicit_update, 
        double * ymom_explicit_update, 
        double * normals, 
        double * areas, 
        double * edgelengths
        )
{
    const int k = 
            threadIdx.x+threadIdx.y*blockDim.x+
            (blockIdx.x+blockIdx.y*gridDim.x)*blockDim.x*blockDim.y;

    int i;
#ifndef REARRANGED_DOMAIN
    int k3=k*3, k6=k*6;
#endif

    double w0, w1, w2, 
           x0, y0, x1, y1, x2, y2,
           avg_h;

    double wx, wy, det,
           hh[3];
    double area, n0, n1, fact;

#ifdef USING_SHARED_MEMORY
    __shared__ double sh_data[ BLOCK_SIZE *6];
#else
    double sidex, sidey;
#endif 


    if (k >= N)
        return;

#ifndef REARRANGED_DOMAIN
    w0 = stage_vertex_values[k3];
    w1 = stage_vertex_values[k3 + 1];
    w2 = stage_vertex_values[k3 + 2];

    x0 = vertex_coordinates[k6];
    y0 = vertex_coordinates[k6 + 1];
    x1 = vertex_coordinates[k6 + 2];
    y1 = vertex_coordinates[k6 + 3];
    x2 = vertex_coordinates[k6 + 4];
    y2 = vertex_coordinates[k6 + 5];
#else
    w0 = stage_vertex_values[k];
    w1 = stage_vertex_values[k + N];
    w2 = stage_vertex_values[k + 2*N];
    

    x0 = vertex_coordinates[k];
    x1 = vertex_coordinates[k + 2*N];
    x2 = vertex_coordinates[k + 4*N];

    
    y0 = vertex_coordinates[k + N];
    y1 = vertex_coordinates[k + 3*N];
    y2 = vertex_coordinates[k + 5*N];
#endif

    //_gradient(x0, y0, x1, y1, x2, y2, w0, w1, w2, &wx, &wy);

    det = (y2 - y0)*(x1 - x0) - (y1 - y0)*(x2 - x0);

    wx = (y2 -y0)*(w1 - w0) - (y1 - y0)*(w2 -w0);
    wx /= det;

    wy = (x1 - x0)*(w2 - w0) - (x2 - x0)*(w1 -w0);
    wy /= det;

    avg_h = stage_centroid_values[k] - bed_centroid_values[k];

    xmom_explicit_update[k] += -g *wx *avg_h;
    ymom_explicit_update[k] += -g *wy *avg_h;

#ifndef REARRANGED_DOMAIN
    hh[0] = stage_edge_values[k3] - bed_edge_values[k3];
    hh[0] *= -0.5 * g * hh[0];
    hh[1] = stage_edge_values[k3+1] - bed_edge_values[k3+1];
    hh[1] *= -0.5 * g * hh[1];
    hh[2] = stage_edge_values[k3+2] - bed_edge_values[k3+2];
    hh[2] *= -0.5 * g * hh[2];
#else
    hh[0] = stage_edge_values[k] - bed_edge_values[k];
    hh[0] *= -0.5 * g * hh[0];
    hh[1] = stage_edge_values[k+N] - bed_edge_values[k+N];
    hh[1] *= -0.5 * g * hh[1];
    hh[2] = stage_edge_values[k+2*N] - bed_edge_values[k+2*N];
    hh[2] *= -0.5 * g * hh[2];
#endif



#ifndef USING_SHARED_MEMORY
    sidex = 0.0;
    sidey = 0.0;
#endif


    area = areas[k];


#ifndef USING_SHARED_MEMORY
    sidex = 0;
    sidey = 0;
#endif


    for ( i = 0 ; i < 3 ; i++ )
    {
#ifndef REARRANGED_DOMAIN
        n0 = normals[k6 + 2*i];
        n1 = normals[k6 + 2*i + 1];

       // fact =  -0.5 * g * hh[i] * hh[i] * edgelengths[k3 + i];
        fact = hh[i] * edgelengths[k3 + i];
#else
        n0 = normals[k + 2*i*N];
        n1 = normals[k + (2*i + 1)*N];

        fact = hh[i] * edgelengths[k + i*N];
#endif


#ifdef USING_SHARED_MEMORY
        sh_data[threadIdx.x + i*blockDim.x] = fact*n0;
        sh_data[threadIdx.x + (i+3)*blockDim.x] = fact*n1;
#else
        sidex += fact*n0;
        sidey += fact*n1;
#endif
    }

#ifdef USING_SHARED_MEMORY
    xmom_explicit_update[k] += -(sh_data[threadIdx.x+3*blockDim.x] + sh_data[threadIdx.x + 4*blockDim.x] + sh_data[threadIdx.x+5*blockDim.x]) / area;
    ymom_explicit_update[k] += -(sh_data[threadIdx.x+3*blockDim.x] + sh_data[threadIdx.x + 4*blockDim.x] + sh_data[threadIdx.x+5*blockDim.x]) / area;
#else
    xmom_explicit_update[k] += -sidex / area;
    ymom_explicit_update[k] += -sidey / area;
#endif
}
